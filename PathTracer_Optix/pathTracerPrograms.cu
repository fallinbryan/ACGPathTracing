#include "hip/hip_runtime.h"
#include <optix.h>


#include "pathTracer.h"
#include "random.h"

#include <sutil/vec_math.h>
#include <cuda/helpers.h>


extern "C" {
  __constant__ PathTraceParams params;
}

struct Onb
{
  __forceinline__ __device__ Onb(const float3& normal)
  {
    m_normal = normal;

    if (fabs(m_normal.x) > fabs(m_normal.z))
    {
      m_binormal.x = -m_normal.y;
      m_binormal.y = m_normal.x;
      m_binormal.z = 0;
    }
    else
    {
      m_binormal.x = 0;
      m_binormal.y = -m_normal.z;
      m_binormal.z = m_normal.y;
    }

    m_binormal = normalize(m_binormal);
    m_tangent = cross(m_binormal, m_normal);
  }

  __forceinline__ __device__ void inverse_transform(float3& p) const
  {
    p = p.x * m_tangent + p.y * m_binormal + p.z * m_normal;
  }

  float3 m_tangent;
  float3 m_binormal;
  float3 m_normal;
};



static __forceinline__ __device__ RadiancePayloadRayData loadClosesthitRadiancePRD()
{
  RadiancePayloadRayData prd = {};

  prd.attenuation.x = __uint_as_float(optixGetPayload_0());
  prd.attenuation.y = __uint_as_float(optixGetPayload_1());
  prd.attenuation.z = __uint_as_float(optixGetPayload_2());
  prd.randomSeed = optixGetPayload_3();
  prd.depth = optixGetPayload_4();
  prd.doneReason = (DoneReason)(optixGetPayload_18());
  return prd;
}

static __forceinline__ __device__ RadiancePayloadRayData loadMissRadiancePRD()
{
  RadiancePayloadRayData prd = {};
  return prd;
}

static __forceinline__ __device__ void storeClosesthitRadiancePRD(RadiancePayloadRayData prd)
{
  optixSetPayload_0(__float_as_uint(prd.attenuation.x));
  optixSetPayload_1(__float_as_uint(prd.attenuation.y));
  optixSetPayload_2(__float_as_uint(prd.attenuation.z));

  optixSetPayload_3(prd.randomSeed);
  optixSetPayload_4(prd.depth);

  optixSetPayload_5(__float_as_uint(prd.emissionColor.x));
  optixSetPayload_6(__float_as_uint(prd.emissionColor.y));
  optixSetPayload_7(__float_as_uint(prd.emissionColor.z));

  optixSetPayload_8(__float_as_uint(prd.radiance.x));
  optixSetPayload_9(__float_as_uint(prd.radiance.y));
  optixSetPayload_10(__float_as_uint(prd.radiance.z));

  optixSetPayload_11(__float_as_uint(prd.origin.x));
  optixSetPayload_12(__float_as_uint(prd.origin.y));
  optixSetPayload_13(__float_as_uint(prd.origin.z));

  optixSetPayload_14(__float_as_uint(prd.direction.x));
  optixSetPayload_15(__float_as_uint(prd.direction.y));
  optixSetPayload_16(__float_as_uint(prd.direction.z));

  optixSetPayload_17(prd.done);
  optixSetPayload_18(prd.doneReason);

}

static __forceinline__ __device__ void storeMissRadiancePRD(RadiancePayloadRayData prd)
{
  optixSetPayload_5(__float_as_uint(prd.emissionColor.x));
  optixSetPayload_6(__float_as_uint(prd.emissionColor.y));
  optixSetPayload_7(__float_as_uint(prd.emissionColor.z));

  optixSetPayload_8(__float_as_uint(prd.radiance.x));
  optixSetPayload_9(__float_as_uint(prd.radiance.y));
  optixSetPayload_10(__float_as_uint(prd.radiance.z));

  optixSetPayload_17(prd.done);
  optixSetPayload_18(prd.doneReason);
}

static __forceinline__ __device__ void cosine_sample_hemisphere(const float u1, const float u2, float3& p)
{
  // Uniformly sample disk.
  const float r = sqrtf(u1);
  const float phi = 2.0f * M_PIf * u2;
  p.x = r * cosf(phi);
  p.y = r * sinf(phi);

  // Project up to hemisphere.
  p.z = sqrtf(fmaxf(0.0f, 1.0f - p.x * p.x - p.y * p.y));
}

static __forceinline__ __device__ bool uniform_sphere_rejection_sample(const float u1, const float u2, const float u3, float3& p)
{
  const float q = sqrtf(u1 * u1 + u2 * u2 + u3 * u3);
  if (q > 1.0f)
    return false;
  else {
    p.x = 2.0f * u1 * q;
    p.y = 2.0f * u2 * q;
    p.z = 2.0f * u3 * q;

    return true;
  
  }
}

static __forceinline__ __device__ const char* doneReasonToString(DoneReason reason)
{
  switch (reason)
  {
  case DoneReason::MISS:
    return "MISS";
  case DoneReason::MAX_DEPTH:
    return "MAX_DEPTH";
  case DoneReason::RUSSIAN_ROULETTE:
    return "RUSSIAN_ROULETTE";
  case DoneReason::NOT_DONE:
    return "NOT_DONE";
  default:
    return "UNKNOWN";
  }
}

static __forceinline__ __device__ bool pixelIsNull(const float3& pixel)
{
  return pixel.x == 0.0f && pixel.y == 0.0f && pixel.z == 0.0f;
}

static __forceinline__ __device__ float safeDivide(float a, float b)
{
  return b == 0.0f ? 0.0f : a / b;
}

static __forceinline__ __device__ float3 safeDivide(float3 a, float b) {
  return make_float3(safeDivide(a.x, b), safeDivide(a.y, b), safeDivide(a.z, b));

}

/**
 * @brief Traces a ray through the scene and updates the payload with the radiance information.
 *
 * This function uses the OptiX API to trace a ray through the scene and gather radiance information.
 * optixTraverse is used to trace the ray and optixInvoke is used to invoke the relavant shader.
 * The relvant shader will update the payload with the radiance and attenuation information.
 *
 * @param handle The handle to the traversable object (scene) to trace the ray through.
 * @param ray_origin The origin of the ray.
 * @param ray_direction The direction of the ray.
 * @param tmin The minimum t value for intersections.
 * @param tmax The maximum t value for intersections.
 * @param prd The payload to store the radiance information in.
 */

// Sample GGX distribution for importance sampling
static __forceinline__ __device__ float3 sampleGGX(float u1, float u2, float roughness, const float3& N)
{
  // Convert (u1, u2) uniform random variables into GGX distribution
  clamp(roughness, 0.001f, 1.0f); // Avoid division by zero (roughness = 0.0f is not allowed
  float phi = 2.0f * M_PIf * u1;
  float cosTheta = sqrtf((1.0f - u2) / (1.0f + (roughness * roughness - 1.0f) * u2));
  float sinTheta = sqrtf(1.0f - cosTheta * cosTheta);

  // Create sample vector in tangent space
  float3 H;
  H.x = sinTheta * cosf(phi);
  H.y = sinTheta * sinf(phi);
  H.z = cosTheta;

  // Transform H to world space
  float3 up = abs(N.z) < 0.999 ? make_float3(0, 0, 1) : make_float3(1, 0, 0);
  float3 tangent = normalize(cross(up, N));
  float3 bitangent = cross(N, tangent);
  float3 sampleDir = H.x * tangent + H.y * bitangent + H.z * N;

  return normalize(sampleDir);
}

// Fresnel-Schlick approximation for conductors derived from the Pbr Book
static __forceinline__ __device__ float3 fresnelSchlickConductor(float cosTheta, float3 eta, float3 k)
{
  float3 eta2 = eta * eta;
  float3 k2 = k * k;

  float3 t1 = eta2 - k2 - make_float3(cosTheta * cosTheta);
  float3 a2plusb2 = make_float3(sqrtf(t1.x * t1.x + 4 * eta2.x * k2.x),
    sqrtf(t1.y * t1.y + 4 * eta2.y * k2.y),
    sqrtf(t1.z * t1.z + 4 * eta2.z * k2.z));

  float3 t2 = a2plusb2 + make_float3(cosTheta * cosTheta);

  float3 Rs = (t2 - 2 * eta * cosTheta + make_float3(cosTheta * cosTheta)) / (t2 + 2 * eta * cosTheta + make_float3(cosTheta * cosTheta));
  float3 Rp = Rs * (t2 - 2 * eta * cosTheta + make_float3(1)) / (t2 + 2 * eta * cosTheta + make_float3(1));

  return (Rs + Rp) * 0.5f;
}

// Fresnel for dialectrics derived from the Pbr Book
static __forceinline__ __device__ float FrDielectric(float cosThetaI, float etaI, float etaT) {
  cosThetaI = clamp(cosThetaI, -1.0f, 1.0f);
  // Flip the interface orientation if the incident ray is inside the material
  bool entering = cosThetaI > 0.0f;
  if (!entering) {
    // Swap etaI and etaT for rays inside the material
    float temp = etaI;
    etaI = etaT;
    etaT = temp;
    cosThetaI = fabs(cosThetaI);
  }

  float sinThetaI = sqrtf(fmaxf(0.0f, 1.0f - cosThetaI * cosThetaI));
  float sinThetaT = etaI / etaT * sinThetaI;

  // Total internal reflection
  if (sinThetaT >= 1.0f) {
    return 1.0f; // When sinThetaT is greater or equal to 1, it indicates total internal reflection.
  }

  float cosThetaT = sqrtf(fmaxf(0.0f, 1.0f - sinThetaT * sinThetaT));

  float rParl = ((etaT * cosThetaI) - (etaI * cosThetaT)) / ((etaT * cosThetaI) + (etaI * cosThetaT));
  float rPerp = ((etaI * cosThetaI) - (etaT * cosThetaT)) / ((etaI * cosThetaI) + (etaT * cosThetaT));
  return (rParl * rParl + rPerp * rPerp) / 2.0f;
}




// GGX/Trowbridge-Reitz Normal Distribution Function
__forceinline__ __device__ float ggxNDF(float cosTheta, float roughness)
{
  clamp(roughness, 0.001f, 1.0f); // Avoid division by zero (roughness = 0.0f is not allowed
  float alpha = roughness * roughness;
  float denom = cosTheta * cosTheta * (alpha * alpha - 1.0f) + 1.0f;
  return (alpha * alpha) / (M_PIf * denom * denom);
}

// Schlick-GGX Geometric Shadowing
__forceinline__ __device__ float geometricSchlickGGX(float NdotV, float roughness)
{
  clamp(roughness, 0.001f, 1.0f); // Avoid division by zero (roughness = 0.0f is not allowed
  float r = (roughness + 1.0f);
  float k = (r * r) / 8.0f; // Beckmann approximation

  float denom = NdotV * (1.0f - k) + k;
  return NdotV / denom;
}

// Combined geometric shadowing for light and view directions
__forceinline__ __device__ float geometricSmith(float NdotV, float NdotL, float roughness)
{
  clamp(roughness, 0.001f, 1.0f); // Avoid division by zero (roughness = 0.0f is not allowed
  float ggxV = geometricSchlickGGX(NdotV, roughness);
  float ggxL = geometricSchlickGGX(NdotL, roughness);
  return ggxV * ggxL;
}

static __forceinline__ __device__ void traceRadiance(
  OptixTraversableHandle handle,
  float3                 ray_origin,
  float3                 ray_direction,
  float                  tmin,
  float                  tmax,
  RadiancePayloadRayData& prd
)
{
  unsigned int u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14, u15, u16, u17, u18;

  u0 = __float_as_uint(prd.attenuation.x);
  u1 = __float_as_uint(prd.attenuation.y);
  u2 = __float_as_uint(prd.attenuation.z);
  u3 = prd.randomSeed;
  u4 = prd.depth;
  u18 = prd.doneReason;

  // Note:
  // This demonstrates the usage of the OptiX shader execution reordering 
  // (SER) API.  In the case of this computationally simple shading code, 
  // there is no real performance benefit.  However, with more complex shaders
  // the potential performance gains offered by reordering are significant.
  optixTraverse(
    RADIANCE_PAYLOAD_TYPE,
    handle,
    ray_origin,
    ray_direction,
    tmin,
    tmax,
    0.0f,                   // rayTime
    OptixVisibilityMask(1),
    OPTIX_RAY_FLAG_NONE,
    0,                      // SBT offset
    NUM_RAYTYPES,           // SBT stride
    0,                      // missSBTIndex
    u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14, u15, u16, u17, u18);

  optixReorder(
    // Application specific coherence hints could be passed in here
  );


  optixInvoke(
    RADIANCE_PAYLOAD_TYPE,
    u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14, u15, u16, u17, u18
  );

  prd.attenuation = make_float3(__uint_as_float(u0), __uint_as_float(u1), __uint_as_float(u2));
  prd.randomSeed = u3;
  prd.depth = u4;

  prd.emissionColor = make_float3(__uint_as_float(u5), __uint_as_float(u6), __uint_as_float(u7));
  prd.radiance = make_float3(__uint_as_float(u8), __uint_as_float(u9), __uint_as_float(u10));
  prd.origin = make_float3(__uint_as_float(u11), __uint_as_float(u12), __uint_as_float(u13));
  prd.direction = make_float3(__uint_as_float(u14), __uint_as_float(u15), __uint_as_float(u16));
  prd.done = u17;
  prd.doneReason = (DoneReason)u18;
}


/**
 * @brief Traces a shadow ray through the scene and returns if the ray is occluded.
 *
 * This function uses the OptiX API to trace a shadow ray through the scene and returns if the ray is occluded.
 * optixTraverse is used to trace the ray and optixHitObjectIsHit is used to check if the ray hit an object.
 *
 * @param handle The handle to the traversable object (scene) to trace the ray through.
 * @param ray_origin The origin of the ray.
 * @param ray_direction The direction of the ray.
 * @param tmin The minimum t value for intersections.
 * @param tmax The maximum t value for intersections.
 * @return true if the ray is occluded, false otherwise.
 */
static __forceinline__ __device__ bool traceOcclusion(
  OptixTraversableHandle handle,
  float3                 ray_origin,
  float3                 ray_direction,
  float                  tmin,
  float                  tmax
)
{
  // We are only casting probe rays so no shader invocation is needed
  optixTraverse(
    handle,
    ray_origin,
    ray_direction,
    tmin,
    tmax, 0.0f,                // rayTime
    OptixVisibilityMask(1),
    OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT | OPTIX_RAY_FLAG_DISABLE_ANYHIT,
    0,                         // SBT offset
    NUM_RAYTYPES,            // SBT stride
    0                          // missSBTIndex
  );
  if (optixHitObjectIsHit()) {
    // get the object that was hit 
    HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();
    if (rt_data->bsdfType == BSDFType::BSDF_REFRACTION) {
      return false;
    }
    else {
      return true;
    }
  }

  return false;
}

/**
 * @brief Generates a ray for the current pixel and traces it through the scene.
 *
 * This function generates a ray for the current pixel and traces it through the scene.
 * The radiance information is accumulated and stored in the frame buffer.
 *
 * The function uses stratified sampling to generate rays for each pixel. Stratified sampling
 * is a method that divides the pixel into smaller sub-pixels and generates a sample within each sub-pixel.
 * This helps to reduce the variance and produce a more accurate image.
 *
 * The function also uses the Russian Roulette method for ray termination. This is a stochastic method
 * used to decide whether to continue or terminate a ray after each bounce. The decision is made based
 * on the intensity of the ray. If the ray's intensity is below a certain threshold, it has a certain
 * probability of being terminated. This helps to reduce the computational cost by not tracing rays
 * that contribute little to the final image.
 *
 * @param params The parameters for the path tracing, including the image width, height, camera parameters, 
 *               and the current frame index.
 */
extern "C" __global__ void __raygen__rg()
{

  const int    w = params.width;
  const int    h = params.height;
  const float3 eye = params.cameraEye;
  const float3 U = params.cameraU;
  const float3 V = params.cameraV;
  const float3 W = params.cameraW;
  const uint3  idx = optixGetLaunchIndex();
  const int    subframe_index = params.currentFrameIdx;
  const unsigned int maxDepth = params.maxDepth;
  

  unsigned int seed = tea<4>(idx.y * w + idx.x, subframe_index);

  float3 result = make_float3(0.0f);
  int i = params.samplesPerPixel;
  RadiancePayloadRayData prd;

  do
  {
    // The center of each pixel is at fraction (0.5,0.5)
    const float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));

    const float2 d = 2.0f * make_float2(
      (static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(w),
      (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(h)
    ) - 1.0f;
    
    float3 ray_direction = normalize(d.x * U + d.y * V + W);
    float3 ray_origin = eye;


    // Initialize the payload for the first time
    prd.attenuation = make_float3(1.f); // The attenuation is initialized to 1
    prd.randomSeed = seed;
    prd.depth = 0;
    prd.doneReason = DoneReason::NOT_DONE;

    for (;; )
    {

      traceRadiance(
        params.handle,
        ray_origin,
        ray_direction,
        0.01f,  // tmin       
        1e16f,  // tmax
        prd
      );

      // at this point the PRD should be updated with the result of the trace
      result += prd.emissionColor;
      result += prd.radiance * prd.attenuation;

      const float p = dot(prd.attenuation, make_float3(0.30f, 0.59f, 0.11f)); // weight by perceived brightness to the human eye

      bool russianRoulette = rnd(prd.randomSeed) > p;

      const bool done = prd.done || russianRoulette || prd.depth >= maxDepth;
      if (done) {
        if(russianRoulette) prd.doneReason = DoneReason::RUSSIAN_ROULETTE;
        if(prd.depth >= maxDepth) prd.doneReason = DoneReason::MAX_DEPTH;
        break;
      }
      prd.attenuation = safeDivide(prd.attenuation, p);

      ray_origin = prd.origin;
      ray_direction = prd.direction;

      ++prd.depth;
    }
  } while (--i);

  const uint3    launch_index = optixGetLaunchIndex();
  const unsigned int image_index = launch_index.y * params.width + launch_index.x;
  float3         accum_color = result / static_cast<float>(params.samplesPerPixel);

  //if (
  //  pixelIsNull(accum_color) && 
  //  subframe_index > 500 && 
  //  prd.doneReason != DoneReason::MISS && 
  //  prd.doneReason != DoneReason::RUSSIAN_ROULETTE && 
  //  prd.doneReason != DoneReason::MAX_DEPTH
  //  )
  //{
  //  //char buffer[256];
  //  //const char* doneReason = doneReasonToString(prd.doneReason);
  //  //strncpy(buffer, doneReason, strlen(doneReason));
  //  printf("Current Depth: %d of %d\n", prd.depth, maxDepth);
  //  printf("Done Reason: %s\n", doneReasonToString(prd.doneReason));
  //  printf("result: %f, %f, %f\n", result.x, result.y, result.z);

  //}

  if (subframe_index > 0)
  {
    const float                 a = 1.0f / static_cast<float>(subframe_index + 1);
    const float3 accum_color_prev = make_float3(params.accumulationBuffer[image_index]);

    //the current frams is linear interpolated with the previous frames
    accum_color = lerp(accum_color_prev, accum_color, a);
  }
  params.accumulationBuffer[image_index] = make_float4(accum_color, 1.0f);
  
  //make_color is helper that clamps and gamma corrects the color into sRGB color space
  params.frameBuffer[image_index] = make_color(accum_color);

}


extern "C" __global__ void __miss__ms()
{

  optixSetPayloadTypes(RADIANCE_PAYLOAD_TYPE);

  MissData* rt_data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
  RadiancePayloadRayData prd = loadMissRadiancePRD();

  prd.radiance = make_float3(rt_data->backgroundColor);
  prd.emissionColor = make_float3(0.f);
  prd.doneReason = DoneReason::MISS;
  prd.done = true;

  storeMissRadiancePRD(prd);
}

extern "C" __global__ void __closesthit__diffuse__ch()
{

  optixSetPayloadTypes(RADIANCE_PAYLOAD_TYPE);

  HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();

  const int    prim_idx = optixGetPrimitiveIndex();
  const float3 ray_dir = optixGetWorldRayDirection();

  const uint3  idx = rt_data->indices[prim_idx];
  const bool useDirectLighting = params.useDirectLighting;
  const bool useImportanceSampling = params.useImportanceSampling;

  const int       prim_idx = optixGetPrimitiveIndex();
  const float3    ray_dir = optixGetWorldRayDirection();

  const uint3     idx = rt_data->indices[prim_idx];
  const bool      useDirectLighting = params.useDirectLighting;
  const float     metallic = rt_data->metallic;
  const float     roughness =  rt_data->roughness;
  const float     IOR = rt_data->IOR;
  const BSDFType  bsdfType = rt_data->bsdfType;


  const float3 v0 = make_float3(rt_data->vertices[idx.x]);
  const float3 v1 = make_float3(rt_data->vertices[idx.y]);
  const float3 v2 = make_float3(rt_data->vertices[idx.z]);


  const float3 N_0 = normalize(cross(v1 - v0, v2 - v0));

  const float3 N = faceforward(N_0, -ray_dir, N_0);
  const float3 P = optixGetWorldRayOrigin() + optixGetRayTmax() * ray_dir;

  RadiancePayloadRayData prd = loadClosesthitRadiancePRD();

  if (prd.depth == 0)
    prd.emissionColor = rt_data->emissionColor;
  else
    prd.emissionColor = make_float3(0.0f);

  unsigned int seed = prd.randomSeed;

  switch (bsdfType)
  {
  case BSDFType::BSDF_DIFFUSE:
  {
    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    float3 w_in;
    bool isRejected = false;
    if (useImportanceSampling)
    {
      cosine_sample_hemisphere(z1, z2, w_in);
      Onb onb(N);
      onb.inverse_transform(w_in);
    }
    else
    {
      float u1;
      float u2;
      float u3;
      do {
        u1 = rnd(seed);
        u2 = rnd(seed);
        u3 = rnd(seed);
      } while (!uniform_sphere_rejection_sample(u1, u2, u3, w_in));
      w_in += N;
      normalize(w_in);
    }

    prd.direction = w_in;
    prd.origin = P;

    prd.attenuation *= rt_data->diffuseColor;
    

    cosine_sample_hemisphere(z1, z2, w_in);
    Onb onb(N);
    onb.inverse_transform(w_in);

    prd.direction = w_in;
    prd.origin = P;
    prd.attenuation *= rt_data->diffuseColor;
    break;
  }
  case BSDFType::BSDF_METALLIC:
  {

    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    float3 microfacetNormal = sampleGGX(z1, z2, roughness, N);
    float3 R = reflect(ray_dir, microfacetNormal); /// Refelction should have a random portion as well

    prd.direction = R;
    prd.origin = P + R * 1e-4f;


    float3 eta = make_float3(1.45, 0.7, 1.55); // Slightly more refraction in the blue channel
    float3 k = make_float3(3.0, 2.2, 3.5); // Higher absorption in the red and blue channels
    float cosTheta = fmaxf(dot(microfacetNormal, -ray_dir), 0.0f);
    float3 F = fresnelSchlickConductor(cosTheta, eta, k);
    float3 F0 = rt_data->diffuseColor;
    float3 color = F * F0;

    prd.attenuation *= color;
    break;

  }
  case BSDFType::BSDF_REFRACTION:
  {

    float3 incidentRayDir = normalize(ray_dir);

    float cos_theta = dot(normalize(-ray_dir), N_0);
    float F = FrDielectric(cos_theta, 1.0f, IOR);


    if (rnd(seed) < F) {
      prd.direction = reflect(incidentRayDir, N_0);

    }
    else {

      float3 refractedDir; // Initialized by the refract function
      bool didRefract = refract(refractedDir, incidentRayDir, N_0, IOR);
      if (didRefract) {
        prd.direction = refractedDir;
      }
      else {
        prd.direction = reflect(incidentRayDir, N_0);
      }
    }
    prd.origin = P + prd.direction * 1e-3f;
    prd.attenuation *= rt_data->diffuseColor;
    break;

  }
  }

  const float z1 = rnd(seed);
  const float z2 = rnd(seed);
  prd.randomSeed = seed;

  float weight = 0.01f;
  AreaLight light = params.areaLight;
  if (useDirectLighting)
  {
    weight = 0.0f;
    //perturb the light position
    const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

    // Calculate properties of light sample (for area based pdf)
    const float  Ldist = length(light_pos - P);
    const float3 L = normalize(light_pos - P);
    const float  nDl = dot(N, L);
    const float  LnDl = -dot(light.normal, L);

        if (nDl > 0.0f && LnDl > 0.0f)
        {
          const bool occluded = traceOcclusion(params.handle,P,L,0.01f, Ldist - 0.01f);  

        if (!occluded)
        {
          const float A = length(cross(light.v1, light.v2));
          weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
          prd.radiance = light.emission * weight;
        }
      }                               
  }
  else {
    
    if (length(rt_data->emissionColor) > 0.0f) {
      prd.radiance = rt_data->emissionColor;
      prd.done = true;
      prd.doneReason = DoneReason::LIGHT_HIT;
    }
    else {
        prd.radiance = make_float3(0.0f);
        prd.done = false;
      }
    
  }


  storeClosesthitRadiancePRD(prd);
}