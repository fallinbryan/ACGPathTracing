#include "hip/hip_runtime.h"
#include <optix.h>


#include "pathTracer.h"
#include "random.h"

#include <sutil/vec_math.h>
#include <cuda/helpers.h>


extern "C" {
  __constant__ PathTraceParams params;
}

struct Onb
{
  __forceinline__ __device__ Onb(const float3& normal)
  {
    m_normal = normal;

    if (fabs(m_normal.x) > fabs(m_normal.z))
    {
      m_binormal.x = -m_normal.y;
      m_binormal.y = m_normal.x;
      m_binormal.z = 0;
    }
    else
    {
      m_binormal.x = 0;
      m_binormal.y = -m_normal.z;
      m_binormal.z = m_normal.y;
    }

    m_binormal = normalize(m_binormal);
    m_tangent = cross(m_binormal, m_normal);
  }

  __forceinline__ __device__ void inverse_transform(float3& p) const
  {
    p = p.x * m_tangent + p.y * m_binormal + p.z * m_normal;
  }

  float3 m_tangent;
  float3 m_binormal;
  float3 m_normal;
};



static __forceinline__ __device__ RadiancePayloadRayData loadClosesthitRadiancePRD()
{
  RadiancePayloadRayData prd = {};

  prd.attenuation.x = __uint_as_float(optixGetPayload_0());
  prd.attenuation.y = __uint_as_float(optixGetPayload_1());
  prd.attenuation.z = __uint_as_float(optixGetPayload_2());
  prd.randomSeed = optixGetPayload_3();
  prd.depth = optixGetPayload_4();
  prd.doneReason = (DoneReason)(optixGetPayload_18());
  return prd;
}

static __forceinline__ __device__ RadiancePayloadRayData loadMissRadiancePRD()
{
  RadiancePayloadRayData prd = {};
  return prd;
}

static __forceinline__ __device__ void storeClosesthitRadiancePRD(RadiancePayloadRayData prd)
{
  optixSetPayload_0(__float_as_uint(prd.attenuation.x));
  optixSetPayload_1(__float_as_uint(prd.attenuation.y));
  optixSetPayload_2(__float_as_uint(prd.attenuation.z));

  optixSetPayload_3(prd.randomSeed);
  optixSetPayload_4(prd.depth);

  optixSetPayload_5(__float_as_uint(prd.emissionColor.x));
  optixSetPayload_6(__float_as_uint(prd.emissionColor.y));
  optixSetPayload_7(__float_as_uint(prd.emissionColor.z));

  optixSetPayload_8(__float_as_uint(prd.radiance.x));
  optixSetPayload_9(__float_as_uint(prd.radiance.y));
  optixSetPayload_10(__float_as_uint(prd.radiance.z));

  optixSetPayload_11(__float_as_uint(prd.origin.x));
  optixSetPayload_12(__float_as_uint(prd.origin.y));
  optixSetPayload_13(__float_as_uint(prd.origin.z));

  optixSetPayload_14(__float_as_uint(prd.direction.x));
  optixSetPayload_15(__float_as_uint(prd.direction.y));
  optixSetPayload_16(__float_as_uint(prd.direction.z));

  optixSetPayload_17(prd.done);
  optixSetPayload_18(prd.doneReason);

}

static __forceinline__ __device__ void storeMissRadiancePRD(RadiancePayloadRayData prd)
{
  optixSetPayload_5(__float_as_uint(prd.emissionColor.x));
  optixSetPayload_6(__float_as_uint(prd.emissionColor.y));
  optixSetPayload_7(__float_as_uint(prd.emissionColor.z));

  optixSetPayload_8(__float_as_uint(prd.radiance.x));
  optixSetPayload_9(__float_as_uint(prd.radiance.y));
  optixSetPayload_10(__float_as_uint(prd.radiance.z));

  optixSetPayload_17(prd.done);
  optixSetPayload_18(prd.doneReason);
}

static __forceinline__ __device__ void cosine_sample_hemisphere(const float u1, const float u2, float3& p)
{
  // Uniformly sample disk.
  const float r = sqrtf(u1);
  const float phi = 2.0f * M_PIf * u2;
  p.x = r * cosf(phi);
  p.y = r * sinf(phi);

  // Project up to hemisphere.
  p.z = sqrtf(fmaxf(0.0f, 1.0f - p.x * p.x - p.y * p.y));
}

static __forceinline__ __device__ bool uniform_sphere_rejection_sample(const float u1, const float u2, const float u3, float3& p)
{
  const float q = sqrtf(u1 * u1 + u2 * u2 + u3 * u3);
  if (q > 1.0f)
    return false;
  else {
    p.x = 2.0f * u1 * q;
    p.y = 2.0f * u2 * q;
    p.z = 2.0f * u3 * q;

    return true;
  
  }
}

static __forceinline__ __device__ const char* doneReasonToString(DoneReason reason)
{
  switch (reason)
  {
  case DoneReason::MISS:
    return "MISS";
  case DoneReason::MAX_DEPTH:
    return "MAX_DEPTH";
  case DoneReason::RUSSIAN_ROULETTE:
    return "RUSSIAN_ROULETTE";
  case DoneReason::NOT_DONE:
    return "NOT_DONE";
  default:
    return "UNKNOWN";
  }
}

static __forceinline__ __device__ bool pixelIsNull(const float3& pixel)
{
  return pixel.x == 0.0f && pixel.y == 0.0f && pixel.z == 0.0f;
}

static __forceinline__ __device__ float safeDivide(float a, float b)
{
  return b == 0.0f ? 0.0f : a / b;
}

static __forceinline__ __device__ float3 safeDivide(float3 a, float b) {
  return make_float3(safeDivide(a.x, b), safeDivide(a.y, b), safeDivide(a.z, b));

}

/**
 * @brief Traces a ray through the scene and updates the payload with the radiance information.
 *
 * This function uses the OptiX API to trace a ray through the scene and gather radiance information.
 * optixTraverse is used to trace the ray and optixInvoke is used to invoke the relavant shader.
 * The relvant shader will update the payload with the radiance and attenuation information.
 *
 * @param handle The handle to the traversable object (scene) to trace the ray through.
 * @param ray_origin The origin of the ray.
 * @param ray_direction The direction of the ray.
 * @param tmin The minimum t value for intersections.
 * @param tmax The maximum t value for intersections.
 * @param prd The payload to store the radiance information in.
 */
static __forceinline__ __device__ void traceRadiance(
  OptixTraversableHandle handle,
  float3                 ray_origin,
  float3                 ray_direction,
  float                  tmin,
  float                  tmax,
  RadiancePayloadRayData& prd
)
{
  unsigned int u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14, u15, u16, u17, u18;

  u0 = __float_as_uint(prd.attenuation.x);
  u1 = __float_as_uint(prd.attenuation.y);
  u2 = __float_as_uint(prd.attenuation.z);
  u3 = prd.randomSeed;
  u4 = prd.depth;
  u18 = prd.doneReason;

  // Note:
  // This demonstrates the usage of the OptiX shader execution reordering 
  // (SER) API.  In the case of this computationally simple shading code, 
  // there is no real performance benefit.  However, with more complex shaders
  // the potential performance gains offered by reordering are significant.
  optixTraverse(
    RADIANCE_PAYLOAD_TYPE,
    handle,
    ray_origin,
    ray_direction,
    tmin,
    tmax,
    0.0f,                   // rayTime
    OptixVisibilityMask(1),
    OPTIX_RAY_FLAG_NONE,
    0,                      // SBT offset
    NUM_RAYTYPES,           // SBT stride
    0,                      // missSBTIndex
    u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14, u15, u16, u17, u18);

  optixReorder(
    // Application specific coherence hints could be passed in here
  );


  optixInvoke(
    RADIANCE_PAYLOAD_TYPE,
    u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14, u15, u16, u17, u18
  );

  prd.attenuation = make_float3(__uint_as_float(u0), __uint_as_float(u1), __uint_as_float(u2));
  prd.randomSeed = u3;
  prd.depth = u4;

  prd.emissionColor = make_float3(__uint_as_float(u5), __uint_as_float(u6), __uint_as_float(u7));
  prd.radiance = make_float3(__uint_as_float(u8), __uint_as_float(u9), __uint_as_float(u10));
  prd.origin = make_float3(__uint_as_float(u11), __uint_as_float(u12), __uint_as_float(u13));
  prd.direction = make_float3(__uint_as_float(u14), __uint_as_float(u15), __uint_as_float(u16));
  prd.done = u17;
  prd.doneReason = (DoneReason)u18;
}


/**
 * @brief Traces a shadow ray through the scene and returns if the ray is occluded.
 *
 * This function uses the OptiX API to trace a shadow ray through the scene and returns if the ray is occluded.
 * optixTraverse is used to trace the ray and optixHitObjectIsHit is used to check if the ray hit an object.
 *
 * @param handle The handle to the traversable object (scene) to trace the ray through.
 * @param ray_origin The origin of the ray.
 * @param ray_direction The direction of the ray.
 * @param tmin The minimum t value for intersections.
 * @param tmax The maximum t value for intersections.
 * @return true if the ray is occluded, false otherwise.
 */
static __forceinline__ __device__ bool traceOcclusion(
  OptixTraversableHandle handle,
  float3                 ray_origin,
  float3                 ray_direction,
  float                  tmin,
  float                  tmax
)
{
  // We are only casting probe rays so no shader invocation is needed
  optixTraverse(
    handle,
    ray_origin,
    ray_direction,
    tmin,
    tmax, 0.0f,                // rayTime
    OptixVisibilityMask(1),
    OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT | OPTIX_RAY_FLAG_DISABLE_ANYHIT,
    0,                         // SBT offset
    NUM_RAYTYPES,            // SBT stride
    0                          // missSBTIndex
  );
  return optixHitObjectIsHit();
}


/**
 * @brief Generates a ray for the current pixel and traces it through the scene.
 *
 * This function generates a ray for the current pixel and traces it through the scene.
 * The radiance information is accumulated and stored in the frame buffer.
 *
 * The function uses stratified sampling to generate rays for each pixel. Stratified sampling
 * is a method that divides the pixel into smaller sub-pixels and generates a sample within each sub-pixel.
 * This helps to reduce the variance and produce a more accurate image.
 *
 * The function also uses the Russian Roulette method for ray termination. This is a stochastic method
 * used to decide whether to continue or terminate a ray after each bounce. The decision is made based
 * on the intensity of the ray. If the ray's intensity is below a certain threshold, it has a certain
 * probability of being terminated. This helps to reduce the computational cost by not tracing rays
 * that contribute little to the final image.
 *
 * @param params The parameters for the path tracing, including the image width, height, camera parameters, 
 *               and the current frame index.
 */
extern "C" __global__ void __raygen__rg()
{
  
  const int    w = params.width;
  const int    h = params.height;
  const float3 eye = params.cameraEye;
  const float3 U = params.cameraU;
  const float3 V = params.cameraV;
  const float3 W = params.cameraW;
  const uint3  idx = optixGetLaunchIndex();
  const int    subframe_index = params.currentFrameIdx;
  const unsigned int maxDepth = params.maxDepth;
  

  unsigned int seed = tea<4>(idx.y * w + idx.x, subframe_index);

  float3 result = make_float3(0.0f);
  int i = params.samplesPerPixel;
  RadiancePayloadRayData prd;

  do
  {
    // The center of each pixel is at fraction (0.5,0.5)
    const float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));

    const float2 d = 2.0f * make_float2(
      (static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(w),
      (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(h)
    ) - 1.0f;
    
    float3 ray_direction = normalize(d.x * U + d.y * V + W);
    float3 ray_origin = eye;


    // Initialize the payload for the first time
    prd.attenuation = make_float3(1.f); // The attenuation is initialized to 1
    prd.randomSeed = seed;
    prd.depth = 0;
    prd.doneReason = DoneReason::NOT_DONE;

    for (;; )
    {

      traceRadiance(
        params.handle,
        ray_origin,
        ray_direction,
        0.01f,  // tmin       
        1e16f,  // tmax
        prd
      );

      // at this point the PRD should be updated with the result of the trace
      result += prd.emissionColor;
      result += prd.radiance * prd.attenuation;

      const float p = dot(prd.attenuation, make_float3(0.30f, 0.59f, 0.11f)); // weight by perceived brightness to the human eye

      bool russianRoulette = rnd(prd.randomSeed) > p;

      const bool done = prd.done || russianRoulette || prd.depth >= maxDepth;
      if (done) {
        if(russianRoulette) prd.doneReason = DoneReason::RUSSIAN_ROULETTE;
        if(prd.depth >= maxDepth) prd.doneReason = DoneReason::MAX_DEPTH;
        break;
      }
      prd.attenuation = safeDivide(prd.attenuation, p);

      ray_origin = prd.origin;
      ray_direction = prd.direction;

      ++prd.depth;
    }
  } while (--i);

  const uint3    launch_index = optixGetLaunchIndex();
  const unsigned int image_index = launch_index.y * params.width + launch_index.x;
  float3         accum_color = result / static_cast<float>(params.samplesPerPixel);

  //if (
  //  pixelIsNull(accum_color) && 
  //  subframe_index > 500 && 
  //  prd.doneReason != DoneReason::MISS && 
  //  prd.doneReason != DoneReason::RUSSIAN_ROULETTE && 
  //  prd.doneReason != DoneReason::MAX_DEPTH
  //  )
  //{
  //  //char buffer[256];
  //  //const char* doneReason = doneReasonToString(prd.doneReason);
  //  //strncpy(buffer, doneReason, strlen(doneReason));
  //  printf("Current Depth: %d of %d\n", prd.depth, maxDepth);
  //  printf("Done Reason: %s\n", doneReasonToString(prd.doneReason));
  //  printf("result: %f, %f, %f\n", result.x, result.y, result.z);

  //}

  if (subframe_index > 0)
  {
    const float                 a = 1.0f / static_cast<float>(subframe_index + 1);
    const float3 accum_color_prev = make_float3(params.accumulationBuffer[image_index]);

    //the current frams is linear interpolated with the previous frames
    accum_color = lerp(accum_color_prev, accum_color, a);
  }
  params.accumulationBuffer[image_index] = make_float4(accum_color, 1.0f);
  
  //make_color is helper that clamps and gamma corrects the color into sRGB color space
  params.frameBuffer[image_index] = make_color(accum_color);

}


extern "C" __global__ void __miss__ms()
{
  
  optixSetPayloadTypes(RADIANCE_PAYLOAD_TYPE);

  MissData* rt_data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
  RadiancePayloadRayData prd = loadMissRadiancePRD();

  prd.radiance = make_float3(rt_data->backgroundColor);
  prd.emissionColor = make_float3(0.f);
  prd.doneReason = DoneReason::MISS;
  prd.done = true;

  storeMissRadiancePRD(prd);
}

extern "C" __global__ void __closesthit__diffuse__ch()
{

  optixSetPayloadTypes(RADIANCE_PAYLOAD_TYPE);

  HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();

  const int    prim_idx = optixGetPrimitiveIndex();
  const float3 ray_dir = optixGetWorldRayDirection();

  const uint3  idx = rt_data->indices[prim_idx];
  const bool useDirectLighting = params.useDirectLighting;
  const bool useImportanceSampling = params.useImportanceSampling;


  const float3 v0 = make_float3(rt_data->vertices[idx.x]);
  const float3 v1 = make_float3(rt_data->vertices[idx.y]);
  const float3 v2 = make_float3(rt_data->vertices[idx.z]);


  const float3 N_0 = normalize(cross(v1 - v0, v2 - v0));

  const float3 N = faceforward(N_0, -ray_dir, N_0);
  const float3 P = optixGetWorldRayOrigin() + optixGetRayTmax() * ray_dir;

  RadiancePayloadRayData prd = loadClosesthitRadiancePRD();

  if (prd.depth == 0)
    prd.emissionColor = rt_data->emissionColor;
  else
    prd.emissionColor = make_float3(0.0f);

  unsigned int seed = prd.randomSeed;
  {
    const float z1 = rnd(seed);
    const float z2 = rnd(seed);


    float3 w_in;
    bool isRejected = false;
    if (useImportanceSampling)
    {
      cosine_sample_hemisphere(z1, z2, w_in);
      Onb onb(N);
      onb.inverse_transform(w_in);
    }
    else
    {
      float u1;
      float u2;
      float u3;
      do {
        u1 = rnd(seed);
        u2 = rnd(seed);
        u3 = rnd(seed);
      } while (!uniform_sphere_rejection_sample(u1, u2, u3, w_in));
      w_in += N;
      normalize(w_in);
    }

    prd.direction = w_in;
    prd.origin = P;

    prd.attenuation *= rt_data->diffuseColor;
    

  }

  const float z1 = rnd(seed);
  const float z2 = rnd(seed);
  prd.randomSeed = seed;

  AreaLight light = params.areaLight;
  float weight = 0.01f;
  if (useDirectLighting) 
  {
    weight = 0.0f;
    const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

    // Calculate properties of light sample (for area based pdf)
    const float  Ldist = length(light_pos - P);
    const float3 L = normalize(light_pos - P);
    const float  nDl = dot(N, L);
    const float  LnDl = -dot(light.normal, L);

    if (nDl > 0.0f && LnDl > 0.0f)
    {
        const bool occluded =
          traceOcclusion(
            params.handle,
            P,
            L,
            0.01f,           // tmin
            Ldist - 0.01f);  // tmax

        if (!occluded)
        {
          const float A = length(cross(light.v1, light.v2));
          weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
          prd.radiance = light.emission * weight;
        }
      }                               
  }
  else {
    
    if (length(rt_data->emissionColor) > 0.0f) {
      prd.radiance = rt_data->emissionColor;
      prd.done = true;
      prd.doneReason = DoneReason::LIGHT_HIT;
    }
    else {
        prd.radiance = make_float3(0.0f);
        prd.done = false;
      }
    
  }


  storeClosesthitRadiancePRD(prd);
}